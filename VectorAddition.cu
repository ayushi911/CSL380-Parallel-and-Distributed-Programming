
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


// CUDA kernel. Each thread takes care of one element of c
// threadIdx.x gives thread id
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
// Get our global thread ID
int id = threadIdx.x;
// Make sure we do not go out of bounds
if (id < n)
c[id] = a[id] + b[id];
}


int main( int argc, char* argv[] ) {
int n = 1000; // Size of vectors
int i;
double *h_a, *h_b; // input vectors
double *h_c; // output vector
size_t bytes = n*sizeof(double); // Size, in bytes, of each vector
// Allocate memory for each vector on host
h_a = (double*)malloc(bytes); h_b = (double*)malloc(bytes); h_c = (double*)malloc(bytes);
// Initialize vectors on host
for( i = 0; i < n; i++ ) { h_a[i] = rand(); h_b[i] = rand(); }
double *d_a, *d_b; // Device input vectors
double *d_c; //Device output vector
// Allocate memory for each vector on GPU
hipMalloc(&d_a, bytes); hipMalloc(&d_b, bytes); hipMalloc(&d_c, bytes);
// Copy data into device (GPU) memory
hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
// Launch kernels
vecAdd<<<1, n>>>(d_a, d_b, d_c, n);
// Copy output data into Host memory
hipMemcpy( d_a, h_a, bytes, hipMemcpyDeviceToHost);
hipMemcpy( d_b, h_b, bytes, hipMemcpyDeviceToHost);
// Free device memory
hipFree(d_a); hipFree(d_b); hipFree(d_c);
// WE ARE DONE – back in Host (CPU) processing
// Free memory
free(h_a); free(h_b); free(h_c);
free(d_a); free(d_b); free(d_c);
return 0;
}
